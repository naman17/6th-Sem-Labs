#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void add(float *a,float *b){
  
  int id = blockIdx.x*blockDim.x+threadIdx.x;
  b[id] = sinf(a[id]);
}

int main(){
  int n;
  float a[10],b[10];

  printf("Enter n:");
  scanf("%d",&n);

  printf("Enter A:\n");
  for(int i=0;i<n;i++)
    scanf("%f",&a[i]);

  float *d_a,*d_b;
  int size = sizeof(float)*n;

  hipMalloc((void**)&d_a,size);
  hipMalloc((void**)&d_b,size);

  hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
  hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);
  
  add<<<n,1>>>(d_a,d_b);

  hipMemcpy(&b,d_b,size,hipMemcpyDeviceToHost);

  for(int i=0;i<n;i++)
    printf("%f ",b[i]);
  printf("\n");
  hipFree(d_a);
  hipFree(d_b);
}