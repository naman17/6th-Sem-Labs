#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void add(int *a,int *b, int *al)
{
  int id=blockIdx.x*blockDim.x+threadIdx.x;
  b[id] = (*al)*a[id] + b[id];
}

int main()
{
  int a[10],b[10],n,al;
  printf("Enter n: ");
  scanf("%d",&n);

  printf("Enter alpha: ");
  scanf("%d",&al);

  printf("Enter X:\n");
  for(int i=0;i<n;i++)
    scanf("%d",&a[i]);

  printf("Enter Y:\n");
  for(int i=0;i<n;i++)
    scanf("%d",&b[i]);

  int *d_a,*d_b,*d_c;
  int size = sizeof(int)*n;

  hipMalloc((void**)&d_a,size);
  hipMalloc((void**)&d_b,size);
  hipMalloc((void**)&d_c,sizeof(int));

  hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
  hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);
  hipMemcpy(d_c,&al,sizeof(int),hipMemcpyHostToDevice);
  
  add<<<n,1>>>(d_a,d_b,d_c);

  hipMemcpy(&b,d_b,size,hipMemcpyDeviceToHost);

  for(int i=0;i<n;i++)
    printf("%d ",b[i]);
  printf("\n");
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
