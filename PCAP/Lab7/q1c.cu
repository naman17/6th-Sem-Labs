#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void add(int *a,int *b,int *c)
{
  int id=blockIdx.x*blockDim.x+threadIdx.x;
  c[id]=a[id]+b[id];
}

int main()
{
  int a[1025],b[1025],c[1025],n;
  printf("Enter n: ");
  scanf("%d",&n);
  printf("Enter A:\n");
  for(int i=0;i<n;i++)
    a[i]=i;
  printf("Enter B:\n");
  for(int i=0;i<n;i++)
    b[i]=i+1;
  int *d_a,*d_b,*d_c;
  int size=sizeof(int)*n;
  hipMalloc((void**)&d_a,size);
  hipMalloc((void**)&d_b,size);
  hipMalloc((void**)&d_c,size);
  hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
  hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);
  add<<<n,256>>>(d_a,d_b,d_c);
  hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);
  for(int i=0;i<n;i++)
    printf("%d ",c[i]);
  printf("\n");
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
