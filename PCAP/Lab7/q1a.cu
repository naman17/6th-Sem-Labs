#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void add(int *a,int *b,int *c)
{
  int id=blockIdx.x*blockDim.x+threadIdx.x;
  c[id]=a[id]+b[id];
}

int main()
{
  int a[10],b[10],c[10],n;
  printf("Enter n: ");
  scanf("%d",&n);
  printf("Enter A:\n");
  for(int i=0;i<n;i++)
    scanf("%d",&a[i]);
  printf("Enter B:\n");
  for(int i=0;i<n;i++)
    scanf("%d",&b[i]);
  int *d_a,*d_b,*d_c;
  int size=sizeof(int)*n;
  hipMalloc((void**)&d_a,size);
  hipMalloc((void**)&d_b,size);
  hipMalloc((void**)&d_c,size);
  hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
  hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);
  add<<<n,1>>>(d_a,d_b,d_c);
  hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);
  for(int i=0;i<n;i++)
    printf("%d ",c[i]);
  printf("\n");
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
