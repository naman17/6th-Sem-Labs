#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void add(int *A, int *B, int *C, int wa, int wb) {
    // Get the 1D Array index of the matrix
    int row = threadIdx.y;
    int col = threadIdx.x;
    int sum = 0;
    for (int i = 0; i < wa; ++i) {
        sum += (A[row*wa + i] * B[i*wb + col]);
    }
    C[row*wb + col] = sum;
}

int main(){
  int a[100], b[100], c[100], n1, m1, n2, m2;
  printf("Enter m1: ");
  scanf("%d",&m1);
  printf("Enter n1: ");
  scanf("%d",&n1);
  printf("Enter Matrix 1:\n");
  for(int i=0;i<n1*m1;i++)
      scanf("%d",&a[i]);

printf("Enter m2: ");
  scanf("%d",&m2);

  if (m2 != n1){
      printf("cannot be multiplied\n");
      exit(0);
  }

  printf("Enter n2: ");
  scanf("%d",&n2);
  printf("Enter Matrix 2:\n");
  for(int i=0;i<n2*m2;i++)
      scanf("%d",&b[i]);

  int *d_a,*d_b,*d_c;
  hipMalloc((void**)&d_a,sizeof(int)*n1*m1);
  hipMalloc((void**)&d_b,sizeof(int)*n2*m2);
  hipMalloc((void**)&d_c,sizeof(int)*m1*n2);

  hipMemcpy(d_a,&a,sizeof(int)*n1*m1,hipMemcpyHostToDevice);
  hipMemcpy(d_b,&b,sizeof(int)*n2*m2,hipMemcpyHostToDevice);
  dim3 block(n2, m1, 1);
  add<<<1, block>>>(d_a, d_b, d_c, n1, n2);
  hipMemcpy(&c,d_c,sizeof(int)*n2*m1,hipMemcpyDeviceToHost);

  for(int i=0;i<m1*n2;i++){
    if (i % n2 == 0)
      printf("\n");
    printf("%d ",c[i]);
  }
  
  printf("\n");
  hipFree(d_a);
  hipFree(d_b);
}
