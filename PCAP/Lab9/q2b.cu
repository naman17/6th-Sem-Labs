#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void add(int *a, int *b, int *c, int m){

  int id=blockIdx.x*blockDim.x+threadIdx.x;
  int offset = 0;
  int n = blockDim.x;
  for (int i = 0; i < m; ++i){
    offset = i*n + id;
    c[offset] =  a[offset] + b[offset];
  }
  
}

int main()
{
  int a[100], b[100], c[100],n, m;
  printf("Enter m: ");
  scanf("%d",&m);
  printf("Enter n: ");
  scanf("%d",&n);
  printf("Enter Matrix 1:\n");
  for(int i=0;i<n*m;i++)
      scanf("%d",&a[i]);

  printf("Enter Matrix 2:\n");
  for(int i=0;i<n*m;i++)
      scanf("%d",&b[i]);

  int *d_a,*d_b,*d_c;
  int size=sizeof(int)*n*m;
  hipMalloc((void**)&d_a,size);
  hipMalloc((void**)&d_b,size);
  hipMalloc((void**)&d_c,size);

  hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
  hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);
  add<<<1, n>>>(d_a, d_b, d_c, m);
  hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);

  for(int i=0;i<n*m;i++){
    if (i % m == 0)
      printf("\n");
    printf("%d ",c[i]);
  }
  
  printf("\n");
  hipFree(d_a);
  hipFree(d_b);
}
