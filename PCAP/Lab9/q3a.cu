#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void add(int *A, int *B, int *C, int wa, int wb) {
    // Get the 1D Array index of the matrix
    int id = threadIdx.x;
    int sum;
    for (int i = 0; i < wb; ++i) {
        sum = 0;
        for (int j = 0; j < wa; ++j){
            sum += (A[id*wa + j] * B[j*wb + i]);
        }
        C[id*wb + i] = sum;
    }
}

int main(){
  int a[100], b[100], c[100], n1, m1, n2, m2;
  printf("Enter m1: ");
  scanf("%d",&m1);
  printf("Enter n1: ");
  scanf("%d",&n1);
  printf("Enter Matrix 1:\n");
  for(int i=0;i<n1*m1;i++)
      scanf("%d",&a[i]);

printf("Enter m2: ");
  scanf("%d",&m2);

  if (m2 != n1){
      printf("cannot be multiplied\n");
      exit(0);
  }

  printf("Enter n2: ");
  scanf("%d",&n2);
  printf("Enter Matrix 2:\n");
  for(int i=0;i<n2*m2;i++)
      scanf("%d",&b[i]);

  int *d_a,*d_b,*d_c;
  hipMalloc((void**)&d_a,sizeof(int)*n1*m1);
  hipMalloc((void**)&d_b,sizeof(int)*n2*m2);
  hipMalloc((void**)&d_c,sizeof(int)*m1*n2);

  hipMemcpy(d_a,&a,sizeof(int)*n1*m1,hipMemcpyHostToDevice);
  hipMemcpy(d_b,&b,sizeof(int)*n2*m2,hipMemcpyHostToDevice);
  add<<<1, m1>>>(d_a, d_b, d_c, n1, n2);
  hipMemcpy(&c,d_c,sizeof(int)*n2*m1,hipMemcpyDeviceToHost);

  for(int i=0;i<m1*n2;i++){
    if (i % n2 == 0)
      printf("\n");
    printf("%d ",c[i]);
  }
  
  printf("\n");
  hipFree(d_a);
  hipFree(d_b);
}
