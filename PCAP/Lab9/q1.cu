#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void add(int *a, int *b, int m){

  int id=blockIdx.x*blockDim.x+threadIdx.x;
  // c[id]=a[id]+b[id];
  // printf("id: %d m: %d ", id, m);
  for (int i = 0; i < m; ++i){
    b[id*m + i] = powf(a[id*m + i], id+1);
    // printf("index %d element %d\n", id*m + i, a[id*m + i]);
  }
  
}

int main()
{
  int a[100], b[100],n, m;
  printf("Enter n: ");
  scanf("%d",&n);
  printf("Enter m: ");
  scanf("%d",&m);
  printf("Enter Matrix:\n");
  for(int i=0;i<n*m;i++)
      scanf("%d",&a[i]);

  int *d_a,*d_b;
  int size=sizeof(int)*n*m;
  hipMalloc((void**)&d_a,size);
  hipMalloc((void**)&d_b,size);

  hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
  add<<<n,1>>>(d_a, d_b, m);
  hipMemcpy(&b,d_b,size,hipMemcpyDeviceToHost);
  for(int i=0;i<n*m;i++){
    if (i % m == 0)
    {
      printf("\n");
    }
      printf("%d ",b[i]);
  }
      

  printf("\n");
  hipFree(d_a);
  hipFree(d_b);
}
