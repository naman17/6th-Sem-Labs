#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void add(int *a, int *b, int *c, int m){

  int id=blockIdx.x*blockDim.x+threadIdx.x;
  c[id] =  a[id] + b[id];
  
}

int main()
{
  int a[100], b[100], c[100],n, m;
  printf("Enter m: ");
  scanf("%d",&m);
  printf("Enter n: ");
  scanf("%d",&n);
  printf("Enter Matrix 1:\n");
  for(int i=0;i<n*m;i++)
      scanf("%d",&a[i]);

  printf("Enter Matrix 2:\n");
  for(int i=0;i<n*m;i++)
      scanf("%d",&b[i]);

  int *d_a,*d_b,*d_c;
  int size=sizeof(int)*n*m;
  hipMalloc((void**)&d_a,size);
  hipMalloc((void**)&d_b,size);
  hipMalloc((void**)&d_c,size);

  hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
  hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);
  add<<<m, n>>>(d_a, d_b, d_c, m);
  hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);

  for(int i=0;i<n*m;i++){
    if (i % m == 0)
      printf("\n");
    printf("%d ",c[i]);
  }
  
  printf("\n");
  hipFree(d_a);
  hipFree(d_b);
}
