#include<stdio.h>
#include<hip/hip_runtime.h>


__device__ int intToBin(int x){
  return 5;
}

__global__ void add(int *A, int *B, int m, int n) {
    int row = threadIdx.y;
    int col = threadIdx.x;

    if ((row%(m-1) == 0) || (col%(n-1) == 0)){
      B[row*n + col] = A[row*n + col];
    }else{
      B[row*n + col] = intToBin(A[row*n + col]);
    }
}

int main(){
  int a[100], b[100], n, m;
  printf("Enter m: ");
  scanf("%d",&m);
  printf("Enter n: ");
  scanf("%d",&n);
  printf("Enter Matrix:\n");
  for(int i=0;i<n*m;i++)
      scanf("%d",&a[i]);

  int *d_a,*d_b;
  int size = sizeof(int)*m*n;
  hipMalloc((void**)&d_a,size);
  hipMalloc((void**)&d_b,size);

  hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
  hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);

  dim3 block(m, n, 1);
  add<<<1, block>>>(d_a, d_b, m, n);

  hipMemcpy(&b,d_b,size,hipMemcpyDeviceToHost);

  for(int i=0;i<n*m;i++){
    if (i % n == 0)
      printf("\n");
    printf("%d ",b[i]);
  }
  
  printf("\n");
  hipFree(d_a);
  hipFree(d_b);
}
