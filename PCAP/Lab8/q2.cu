#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<>
#include<string.h>

__global__ void reverse( char *ip, int len){
    // Get the index of the current element to be processed
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    int start = 0;
    int end = 0;
  for(int j=0;j<len;++j) {
    if(ip[j] == ' ' || ip[j] == '\0'){
      i--;
      if(i == 0) start = j+1;
      if(i < 0){
        end = j-1;
        break;
      }
    }
  }
  while(start < end){
    i = ip[start];
    ip[start++] = ip[end];
    ip[end--] = i;
  }
}

int main(){
  int n, len;
  char a[100],b[100];

  printf("Enter string:\n");
  gets(a);
  printf("Enter n:\n");
  scanf("%d", &n);

  len = strlen(a);
  a[len++] = ' ';
  a[len] = '\0';

  char *d_a;
  memset(b, 0, 100);

  hipMalloc((void**)&d_a,len);

  hipMemcpy(d_a,&a,len,hipMemcpyHostToDevice);
  
  reverse<<<n,1>>>(d_a, len);

  hipMemcpy(&b,d_a,len,hipMemcpyDeviceToHost);

  printf("%s\n", b);
  hipFree(d_a);
}